
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>


void initialize_matrix(int *mtx, int m, int n) {
    for(int i = 0; i< m*n ; ++i) {
        mtx[i] = rand()%10;   
    }
}

void cpu_matrix_mul(int *A, int *B, int* C, int row, int width, int col) {
    for(int i = 0; i< row; ++i){
        for(int j = 0; j< col; ++j) {
            int value = 0;
            for(int k = 0; k < width ; ++k) {
                value += A[i * width + k] * B[k * col + j];
            }
            C[i * col + j] = value;
        }
    }
}


int main() {
    
    int row = 1 << 13;
    int width = 1 << 12;
    int col = 1 << 11;

    int *hA, *hB, *hC;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipHostMalloc(&hA, row * width * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&hB, width * col * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&hC, row * col * sizeof(int), hipHostMallocDefault);

    initialize_matrix(hA, row, width);
    initialize_matrix(hB, width, col);


    hipEventRecord(start, 0);
    cpu_matrix_mul(hA, hB, hC, row, width, col);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float gpu_elapsed_time_ms;
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    
    printf("Time Elapsed on GPU Matrix Multiplication: %f\n", gpu_elapsed_time_ms);

    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    return 0;
}


#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <cassert>

#define TILE_WIDTH 16

__global__ void gpu_matrix_mult(int *A, int *B, int *C, int row, int width, int col) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if(c < row && c < col) {
        for(int i = 0; i < width ; ++i) {
            sum += A[r * width + i] * B[i * col + c];
        }
        C[r * col + c] = sum;
    }    
}

void initialize_matrix(int *mtx, int m, int n) {
    for(int i = 0; i< m*n ; ++i) {
        mtx[i] = rand()%10;   
    }
}
int main() {
    
    int row = 1 << 13;
    int width = 1 << 12;
    int col = 1 << 11;

    int *hA, *hB, *hC;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipHostMalloc(&hA, row * width * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&hB, width * col * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&hC, row * col * sizeof(int), hipHostMallocDefault);

    initialize_matrix(hA, row, width);
    initialize_matrix(hB, width, col);

    int *deviceA, *deviceB, *deviceC;
    hipMalloc((void**)&deviceA, sizeof(int) * row * width);
    hipMalloc((void**)&deviceB, sizeof(int) * col * width);
    hipMalloc((void**)&deviceC, sizeof(int) * row * col);
    
    hipMemcpy(deviceA, hA, sizeof(int) * row * width, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hB, sizeof(int) * width * col, hipMemcpyHostToDevice);
    
    int threadX = 32;
    int threadY = 16;
    int grid_rows = (row + threadY -1)/threadY;
    int grid_cols = (col + threadX -1)/threadX;

    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(threadX, threadY);

    hipEventRecord(start, 0);
    gpu_matrix_mult<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, row, width, col);
    hipMemcpy(hC, deviceC, sizeof(int) * row * col, hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float gpu_elapsed_time_ms;
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);

    printf("Time Elapsed on GPU Matrix Multiplication: %f\n", gpu_elapsed_time_ms);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
    hipHostFree(hA);
    hipHostFree(hB);
    hipHostFree(hC);
    return 0;
}
